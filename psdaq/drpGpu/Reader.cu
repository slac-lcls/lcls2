#include "hip/hip_runtime.h"
#include "Reader.hh"

#include "Detector.hh"
#include "drp/spscqueue.hh"
#include "psalg/utils/SysLog.hh"
#include "psdaq/service/EbDgram.hh"
#include "psdaq/service/kwargs.hh"
#include "psdaq/trigger/TriggerPrimitive.hh"

#include <thread>

using logging = psalg::SysLog;
using namespace XtcData;
using namespace Pds;
using namespace Drp;
using namespace Drp::Gpu;


Reader::Reader(unsigned                 panel,
               const Parameters&        para,
               MemPoolGpu&              pool,
               Detector&                det,
               size_t                   trgPrimitiveSize,
               const cuda::atomic<int>& terminate_d) :
  m_pool         (pool),
  m_det          (det),
  m_terminate_d  (terminate_d),
  m_panel        (panel),
  m_para         (para)
{
  //printf("*** Reader ctor 1\n");

  // Set up buffer index allocator for DMA to Collector comms
  m_readerQueue.h = new Gpu::RingIndexDtoD(m_pool.nbuffers(), m_pool.dmaCount(), m_terminate_d);
  chkError(hipMalloc(&m_readerQueue.d,                  sizeof(*m_readerQueue.d)));
  chkError(hipMemcpy( m_readerQueue.d, m_readerQueue.h, sizeof(*m_readerQueue.d), hipMemcpyHostToDevice));

  // Allocate a stream per buffer
  m_streams.resize(m_pool.dmaCount());
  for (auto& stream : m_streams) {
    chkFatal(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  }
  //printf("*** Reader ctor 2, %zu streams\n", m_streams.size());

  // Set up a done flag to cache m_terminate's value and avoid some PCIe transactions
  chkError(hipMalloc(&m_done,    sizeof(*m_done)));
  chkError(hipMemset( m_done, 0, sizeof(*m_done)));
  //printf("*** Reader ctor 3\n");

  // Keep track of the head index of each Reader stream
  for (unsigned i = 0; i < m_pool.dmaCount(); ++i) {
    chkError(hipMalloc(&m_head[i],    sizeof(*m_head[i])));
    chkError(hipMemset( m_head[i], 0, sizeof(*m_head[i])));
  }
  //printf("*** Reader ctor 4: trgPrmtv sz %zu\n", trgPrimitiveSize);

  // Prepare buffers visible to the host for receiving headers
  const size_t bufSz = sizeof(DmaDsc)+sizeof(TimingHeader) + trgPrimitiveSize;
  //printf("*** Reader ctor 4a: sz %zu, nbufs %u\n", bufSz, m_pool.nbuffers());
  m_pool.createHostBuffers(panel, bufSz);
  //printf("*** Reader ctor 5\n");

  // Prepare the CUDA graphs
  m_graphExecs.resize(m_pool.dmaCount());
  for (unsigned i = 0; i < m_pool.dmaCount(); ++i) {
    if (_setupGraphs(i)) {
      logging::critical("Failed to set up Reader[%u] graphs", panel);
      abort();
    }
  }
  //printf("*** Reader ctor 6\n");
}

Reader::~Reader()
{
  printf("*** Reader dtor 1\n");
  for (auto& graphExec : m_graphExecs) {
    chkError(hipGraphExecDestroy(graphExec));
  }
  printf("*** Reader dtor 2\n");

  m_pool.destroyHostBuffers(m_panel);
  printf("*** Reader dtor 3\n");

  for (unsigned i = 0; i < m_pool.dmaCount(); ++i) {
    chkError(hipFree(m_head[i]));
  }
  chkError(hipFree(m_done));
  printf("*** Reader dtor 4\n");

  for (auto& stream : m_streams) {
    chkError(hipStreamDestroy(stream));
  }
  printf("*** Reader dtor 5\n");

  if (m_readerQueue.d)  chkError(hipFree(m_readerQueue.d));
  delete m_readerQueue.h;
  printf("*** Reader dtor 6\n");
}

int Reader::_setupGraphs(unsigned instance)
{
  hipGraph_t      graph;
  hipGraphExec_t& graphExec = m_graphExecs[instance];
  hipStream_t     stream    = m_streams[instance];

  // Generate the graph
  //printf("*** Reader setupGraphs 1.%u\n", instance);
  auto& hostWriteBufs = m_pool.hostWrtBufsVec_h()[m_panel][0]; // [0] is the base ptr of the whole allocation
  //printf("*** Reader setupGraphs 2.%u\n", instance);
  // Attach host-visible memory to the stream
  chkError(hipStreamAttachMemAsync(stream, hostWriteBufs, 0, hipMemAttachHost));
  //printf("*** Reader setupGraphs 3.%u\n", instance);
  logging::debug("Recording Reader graph %d", instance);
  const auto& panel = m_pool.panels()[m_panel];
  graph = _recordGraph(instance, panel.dmaBuffers[instance].dptr, panel.hwWriteStart);
  //printf("*** Reader setupGraphs 4.%u\n", instance);
  if (graph == 0) {
    return -1;
  }

  // Instantiate the graph. The resulting hipGraphExec_t may only be executed once
  // at any given time.  I believe it can be reused, but it cannot be launched
  // while it is already running.  If we wanted to launch multiple, we would
  // instantiate multiple hipGraphExec_t's and then launch those individually.
  if (chkError(hipGraphInstantiate(&graphExec, graph, hipGraphInstantiateFlagDeviceLaunch),
               "Reader graph create failed")) {
    return -1;
  }
  //printf("*** Reader setupGraphs 5.%u\n", instance);

  // No need to hang on to the stream info
  hipGraphDestroy(graph);

  // Upload the graph so it can be launched by the scheduler kernel later
  logging::debug("Uploading Reader graph %u...", instance);
  if (chkError(hipGraphUpload(graphExec, stream), "Reader graph upload failed")) {
    return -1;
  }
  //printf("*** Reader setupGraphs 6.%u\n", instance);

  return 0;
}

// Wait for the DMA size word to become non-zero
static __global__ void _waitForDMA(const volatile uint32_t* __restrict__ mem,
                                   unsigned                              instance,
                                   Gpu::RingIndexDtoD&                   readerQueue,
                                   unsigned*                __restrict__ head,
                                   const cuda::atomic<int>&              terminate,
                                   bool*                    __restrict__ done)
{
  //printf("### Reader waitForDMA 1.%u\n", instance);

  // Allocate the index of the next set of intermediate buffers to be used
  *head = readerQueue.prepare(instance);
  //printf("### Reader waitForDMA 1.%u, idx %u\n", instance, *head);

  // Wait for data to be DMAed
  //printf("### Reader waitForDMA 2.%u, mem %p\n", instance, mem);
  while (*mem == 0) {
    if ( (*done = terminate.load(cuda::memory_order_acquire)) )  break;
  }
  //printf("### Reader waitForDMA 3.%u, *mem %08x, done = %u\n", instance, *mem, *done);
}

// This copies the DmaDsc and TimingHeader into a host-visible buffer
static __global__ void _event(uint32_t** const __restrict__ outBufs,
                              uint32_t*  const __restrict__ in,
                              unsigned                      instance,
                              const unsigned&               idx,
                              const bool&                   done)
{
  //printf("### Reader _event 1.%u, done %d, idx %u\n", instance, done, idx);
  if (done)  return;
  //printf("### Reader _event 1.%ua, done %d, idx %u, outBufs %p\n", instance, done, idx, outBufs);
  //printf("### Reader _event 1.%ub, done %d, idx %u, *outBufs %p\n", instance, done, idx, *outBufs);

  uint32_t* const __restrict__ out = outBufs[idx];
  //if (threadIdx.x == 0)  printf("### Reader _event 1.%uc, done %d, idx %u, pid %08x%08x\n", instance, done, idx, in[9], in[8]);

  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  auto nWords = (sizeof(DmaDsc)+sizeof(TimingHeader))/sizeof(*out);
  //printf("### Reader _event 2.%u, offset %d, nWords %lu\n", instance, offset, nWords);
  for (int i = offset; i < nWords; i += blockDim.x * gridDim.x) {
    out[i] = in[i];
  }
  //printf("### Reader _event 3.%u\n", instance);
}

// This will re-launch the current graph
static __global__ void _graphLoop(const unsigned&     idx,
                                  Gpu::RingIndexDtoD& readerQueue,
                                  const bool&         done)
{
  //printf("### Reader graphLoop 1, done %d, idx %u\n", done, idx);
  if (done)  return;
  //printf("### Reader graphLoop 1a, idx %u\n", idx);

  readerQueue.produce(idx);
  //printf("### Reader graphLoop 2, idx %u\n", idx);

  hipGraphLaunch(cudaGetCurrentGraphExec(), cudaStreamGraphTailLaunch);
  //printf("### Reader graphLoop 3\n");
}

/******************************************************************************
 * Records a CUDA graph for later instantiation and execution.
 * The nodes within a CUDA graph define the execution steps of what amounts to
 * a "command buffer", in traditional graphics terms.  Edges between nodes on
 * the graph define dependencies.  The execution flow of the GPU DRP
 * application can be accurately described using the graph structure as defined
 * by the CUDA graph API.  Normally, CUDA API calls that run on the GPU (i.e.
 * cuStreamWriteXXX) are converted into an internal representation and inserted
 * into a command buffer within the CUDA driver.  The sync functions can then
 * be used to describe dependencies between steps, however this involves the
 * host and thus introduces latency between steps.  In the case of CUDA graphs,
 * we can avoid host involvement completely and simply give the GPU a list of
 * instructions to execute.  We can even tell the GPU to launch new graphs on
 * its own, if we wanted to cut host involvement out entirely.
 ******************************************************************************/
hipGraph_t Reader::_recordGraph(unsigned    instance,
                                 hipDeviceptr_t dmaBuffer,
                                 hipDeviceptr_t hwWriteStart)
{
  //printf("*** Reader record 1.%d\n", instance);
  auto stream          = m_streams[instance];
  auto hostWriteBufs_d = m_pool.hostWrtBufsVec_d()[m_panel];
  auto hostWriteBufsSz = m_pool.hostWrtBufsSize();
  //printf("*** Reader record 2.%d, hostWriteBufs_d[%u] %p, sz %zu\n",
  //       instance, m_panel, hostWriteBufs_d, hostWriteBufsSz);

  if (chkError(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal),
               "Stream begin-capture failed")) {
    return 0;
  }

  /****************************************************************************
   * Clear the handshake space
   * Originally was hipStreamWriteValue32, but the stream functions are not
   * supported within graphs. hipMemsetD32Async acts as a good replacement.
   ****************************************************************************/
  chkError(hipMemsetD32Async(dmaBuffer + 4, 0, 1, stream));

  // Wipe the buffer (for debugging; normally commented out for performance)
  //chkError(hipMemsetD32Async(dmaBuffer, 0, m_pool.dmaSize() / 4, stream));

#ifndef HOST_REARMS_DMA
  // Write to the DMA start register in the FPGA to trigger the write
  chkError(hipMemsetD8Async(hwWriteStart + 4 * instance, 1, 1, stream));
#endif

  /*****************************************************************************
   * Spin on the handshake location until the value is non-zero
   * This waits for the data to arrive before starting the processing
   * Originally this was a call to cuStreamWait, but that is not supported by
   * graphs, so instead we use a waitForDMA kernel to spin on the location
   * until data is ready to be processed.
   * @todo: This may have negative implications on GPU scheduling.  Profile it!
   ****************************************************************************/
  _waitForDMA<<<1, 1, 1, stream>>>((uint32_t*)(dmaBuffer + 4),
                                   instance,
                                   *m_readerQueue.d,
                                   m_head[instance],
                                   m_terminate_d,
                                   m_done);
  //printf("*** Reader record 3.%d\n", instance);

  // Copy the DMA descriptor and the timing header to host-visible managed memory buffers
  constexpr auto iPayload { (sizeof(DmaDsc)+sizeof(TimingHeader))/sizeof(uint32_t) };
  _event<<<1, iPayload, 0, stream>>>(hostWriteBufs_d, (uint32_t*)dmaBuffer, instance, *m_head[instance], *m_done);
  //printf("*** Reader record 4.%d\n", instance);

  // Calibrate the raw data from the DMA buffers into the calibrated data buffers
  m_det.recordGraph(stream, *m_head[instance], m_panel, (uint16_t*)(dmaBuffer + iPayload));
  //printf("*** Reader record 5.%d\n", instance);

  // Publish the current head index and re-launch
  _graphLoop<<<1, 1, 0, stream>>>(*m_head[instance], *m_readerQueue.d, *m_done);
  //printf("*** Reader record 6.%d\n", instance);

  hipGraph_t graph;
  if (chkError(hipStreamEndCapture(stream, &graph), "Stream end-capture failed")) {
    return 0;
  }

  return graph;
}

void Reader::start()
{
  logging::info("Reader[%d] starting", m_panel);
  chkError(hipCtxSetCurrent(m_pool.context().context()));  // Needed, else kernels misbehave

  const auto& panel = m_pool.panels()[m_panel];

  // Ensure that timing messages are DMAed to the GPU
  dmaTgtSet(panel.gpu, DmaTgt_t::GPU);
  //printf("*** Reader[%d] start 1\n", m_panel);

  // Ensure that the DMA round-robin index starts with buffer 0
  dmaIdxReset(panel.gpu);
  //printf("*** Reader[%d] start 2\n", m_panel);

#ifdef HOST_REARMS_DMA
  // Write to the DMA start register in the FPGA
  for (unsigned dmaIdx = 0; dmaIdx < m_pool.dmaCount(); ++dmaIdx) {
    //printf("*** Reader[%d] start: Enable write to DMA buffer %u\n", m_panel, dmaIdx);
    auto rc = gpuSetWriteEn(panel.gpu.fd(), dmaIdx);
    if (rc < 0) {
      logging::critical("Failed to reenable buffer %u for write: %zd: %m", dmaIdx, rc);
      abort();
    }
  }
#endif
  //printf("*** Reader[%d] start 3\n", m_panel);

  // Launch the DMA graphs
  for (unsigned dmaIdx = 0; dmaIdx < m_pool.dmaCount(); ++dmaIdx) {
    chkFatal(hipGraphLaunch(m_graphExecs[dmaIdx], m_streams[dmaIdx]));
  }
  //printf("*** Reader[%d] start 4\n", m_panel);
}
