#include "hip/hip_runtime.h"
#include "Reader.hh"

#include "Detector.hh"
#include "drp/spscqueue.hh"
#include "psalg/utils/SysLog.hh"
#include "psdaq/service/EbDgram.hh"
#include "psdaq/service/kwargs.hh"
#include "psdaq/trigger/TriggerPrimitive.hh"

#include <thread>

using logging = psalg::SysLog;
using namespace XtcData;
using namespace Pds;
using namespace Drp;
using namespace Drp::Gpu;


Reader::Reader(unsigned                     panel,
               const Parameters&            para,
               MemPoolGpu&                  pool,
               Detector&                    det,
               size_t                       trgPrimitiveSize,
               const cuda::atomic<uint8_t>& terminate_d) :
  m_pool       (pool),
  m_det        (det),
  m_terminate_d(terminate_d),
  m_panel      (panel),
  m_para       (para)
{
  // Set up buffer index allocator for DMA to Collector comms
  m_readerQueue.h = new Gpu::RingIndexDtoD(m_pool.nbuffers(), m_pool.dmaCount(), m_terminate_d);
  chkError(hipMalloc(&m_readerQueue.d,                  sizeof(*m_readerQueue.d)));
  chkError(hipMemcpy( m_readerQueue.d, m_readerQueue.h, sizeof(*m_readerQueue.d), hipMemcpyHostToDevice));

  // Allocate a stream per buffer
  m_streams.resize(m_pool.dmaCount());
  for (auto& stream : m_streams) {
    chkFatal(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  }

  // Keep track of the head index of each Reader stream
  for (unsigned i = 0; i < m_pool.dmaCount(); ++i) {
    chkError(hipMalloc(&m_head[i],    sizeof(*m_head[i])));
    chkError(hipMemset( m_head[i], 0, sizeof(*m_head[i])));
  }

  // Prepare buffers visible to the host for receiving headers
  const size_t bufSz = sizeof(DmaDsc)+sizeof(TimingHeader) + trgPrimitiveSize;
  m_pool.createHostBuffers(panel, bufSz);

  // Prepare the CUDA graphs
  m_graphExecs.resize(m_pool.dmaCount());
  for (unsigned i = 0; i < m_pool.dmaCount(); ++i) {
    if (_setupGraphs(i)) {
      logging::critical("Failed to set up Reader[%u] graphs", panel);
      abort();
    }
  }
}

Reader::~Reader()
{
  for (auto& graphExec : m_graphExecs) {
    chkError(hipGraphExecDestroy(graphExec));
  }

  m_pool.destroyHostBuffers(m_panel);

  for (unsigned i = 0; i < m_pool.dmaCount(); ++i) {
    chkError(hipFree(m_head[i]));
  }

  for (auto& stream : m_streams) {
    chkError(hipStreamDestroy(stream));
  }

  if (m_readerQueue.d)  chkError(hipFree(m_readerQueue.d));
  delete m_readerQueue.h;
}

int Reader::_setupGraphs(unsigned instance)
{
  hipGraph_t      graph;
  hipGraphExec_t& graphExec = m_graphExecs[instance];
  hipStream_t     stream    = m_streams[instance];

  // Generate the graph
  logging::debug("Recording Reader graph %d", instance);
  const auto& panel = m_pool.panels()[m_panel];
  graph = _recordGraph(instance, panel.dmaBuffers[instance].dptr, panel.hwWriteStart);
  if (graph == 0) {
    return -1;
  }

  // Instantiate the graph. The resulting hipGraphExec_t may only be executed once
  // at any given time.  I believe it can be reused, but it cannot be launched
  // while it is already running.  If we wanted to launch multiple, we would
  // instantiate multiple hipGraphExec_t's and then launch those individually.
  if (chkError(hipGraphInstantiate(&graphExec, graph, hipGraphInstantiateFlagDeviceLaunch),
               "Reader graph create failed")) {
    return -1;
  }

  // No need to hang on to the stream info
  hipGraphDestroy(graph);

  // Upload the graph so it can be launched by the scheduler kernel later
  logging::debug("Uploading Reader graph %u...", instance);
  if (chkError(hipGraphUpload(graphExec, stream), "Reader graph upload failed")) {
    return -1;
  }

  return 0;
}

// Wait for the DMA size word to become non-zero
static __global__ void _waitForDMA(const volatile uint32_t* __restrict__ mem,
                                   unsigned                              instance,
                                   Gpu::RingIndexDtoD&                   readerQueue,
                                   unsigned*                __restrict__ head,
                                   const cuda::atomic<uint8_t>&          terminate)
{
  // Allocate the index of the next set of intermediate buffers to be used
  *head = readerQueue.prepare(instance);

  // Wait for data to be DMAed
  while (*mem == 0) {
    if (terminate.load(cuda::memory_order_acquire))  break;
    //__nanosleep(5000);                  // Suspend the thread
  }
}

// This copies the DmaDsc and TimingHeader into a host-visible buffer
static __global__ void _event(uint32_t* const __restrict__ outBufs,
                              const size_t                 outBufsCnt,
                              uint32_t* const __restrict__ in,
                              unsigned                     instance,
                              const unsigned&              idx,
                              const cuda::atomic<uint8_t>& terminate)
{
  if (terminate.load(cuda::memory_order_acquire))  return;

  uint32_t* const __restrict__ out = outBufs + idx * outBufsCnt;
  //if (threadIdx.x == 0)  printf("### Reader::_event: pnl %u, idx %u, out %p\n", instance, idx, out);

  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  constexpr auto count = (sizeof(DmaDsc)+sizeof(TimingHeader))/sizeof(*out);
  for (int i = offset; i < count; i += blockDim.x * gridDim.x) {
    out[i] = in[i];
  }
}

// This will re-launch the current graph
static __global__ void _graphLoop(const unsigned&              idx,
                                  Gpu::RingIndexDtoD&          readerQueue,
                                  const cuda::atomic<uint8_t>& terminate)
{
  if (terminate.load(cuda::memory_order_acquire))  return;

  readerQueue.produce(idx);

  hipGraphLaunch(cudaGetCurrentGraphExec(), cudaStreamGraphTailLaunch);
}

/******************************************************************************
 * Records a CUDA graph for later instantiation and execution.
 * The nodes within a CUDA graph define the execution steps of what amounts to
 * a "command buffer", in traditional graphics terms.  Edges between nodes on
 * the graph define dependencies.  The execution flow of the GPU DRP
 * application can be accurately described using the graph structure as defined
 * by the CUDA graph API.  Normally, CUDA API calls that run on the GPU (i.e.
 * cuStreamWriteXXX) are converted into an internal representation and inserted
 * into a command buffer within the CUDA driver.  The sync functions can then
 * be used to describe dependencies between steps, however this involves the
 * host and thus introduces latency between steps.  In the case of CUDA graphs,
 * we can avoid host involvement completely and simply give the GPU a list of
 * instructions to execute.  We can even tell the GPU to launch new graphs on
 * its own, if we wanted to cut host involvement out entirely.
 ******************************************************************************/
hipGraph_t Reader::_recordGraph(unsigned    instance,
                                 hipDeviceptr_t dmaBuffer,
                                 hipDeviceptr_t hwWriteStart)
{
  auto stream         = m_streams[instance];
  auto hostWrtBufs_d  = m_pool.hostWrtBufsVec_d()[m_panel];
  auto hostWrtBufsCnt = m_pool.hostWrtBufsSize() / sizeof(*hostWrtBufs_d);

  if (chkError(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal),
               "Stream begin-capture failed")) {
    return 0;
  }

  /****************************************************************************
   * Clear the handshake space
   * Originally was hipStreamWriteValue32, but the stream functions are not
   * supported within graphs. hipMemsetD32Async acts as a good replacement.
   ****************************************************************************/
  chkError(hipMemsetD32Async(dmaBuffer + 4, 0, 1, stream));

  // Wipe the buffer (for debugging; normally commented out for performance)
  //chkError(hipMemsetD32Async(dmaBuffer, 0, m_pool.dmaSize() / 4, stream));

#ifndef HOST_REARMS_DMA
  // Write to the DMA start register in the FPGA to trigger the write
  chkError(hipMemsetD8Async(hwWriteStart + 4 * instance, 1, 1, stream));
#endif

  /*****************************************************************************
   * Spin on the handshake location until the value is non-zero
   * This waits for the data to arrive before starting the processing
   * Originally this was a call to cuStreamWait, but that is not supported by
   * graphs, so instead we use a waitForDMA kernel to spin on the location
   * until data is ready to be processed.
   * @todo: This may have negative implications on GPU scheduling.  Profile it!
   ****************************************************************************/
  _waitForDMA<<<1, 1, 1, stream>>>((uint32_t*)(dmaBuffer + 4),
                                   instance,
                                   *m_readerQueue.d,
                                   m_head[instance],
                                   m_terminate_d);

  // Copy the DMA descriptor and the timing header to host-visible pinned memory buffers
  constexpr auto iPayload { (sizeof(DmaDsc)+sizeof(TimingHeader))/sizeof(uint32_t) };
  _event<<<1, iPayload, 0, stream>>>(hostWrtBufs_d,
                                     hostWrtBufsCnt,
                                     (uint32_t*)dmaBuffer,
                                     instance,
                                     *m_head[instance],
                                     m_terminate_d);

  // Calibrate the raw data from the DMA buffers into the calibrated data buffers
  m_det.recordGraph(stream, *m_head[instance], m_panel, (uint16_t*)(dmaBuffer + iPayload));

  // Publish the current head index and re-launch
  _graphLoop<<<1, 1, 0, stream>>>(*m_head[instance], *m_readerQueue.d, m_terminate_d);

  hipGraph_t graph;
  if (chkError(hipStreamEndCapture(stream, &graph), "Stream end-capture failed")) {
    return 0;
  }

  return graph;
}

void Reader::start()
{
  logging::info("Reader[%d] starting", m_panel);
  chkError(hipCtxSetCurrent(m_pool.context().context()));  // Needed, else kernels misbehave

  const auto& panel = m_pool.panels()[m_panel];

  // Ensure that timing messages are DMAed to the GPU
  dmaTgtSet(panel.gpu, DmaTgt_t::GPU);

  // Ensure that the DMA round-robin index starts with buffer 0
  dmaIdxReset(panel.gpu);

#ifdef HOST_REARMS_DMA
  // Write to the DMA start register in the FPGA
  for (unsigned dmaIdx = 0; dmaIdx < m_pool.dmaCount(); ++dmaIdx) {
    auto rc = gpuSetWriteEn(panel.gpu.fd(), dmaIdx);
    if (rc < 0) {
      logging::critical("Failed to reenable buffer %u for write: %zd: %m", dmaIdx, rc);
      abort();
    }
  }
#endif

  // Launch the DMA graphs
  for (unsigned dmaIdx = 0; dmaIdx < m_pool.dmaCount(); ++dmaIdx) {
    chkFatal(hipGraphLaunch(m_graphExecs[dmaIdx], m_streams[dmaIdx]));
  }
}
