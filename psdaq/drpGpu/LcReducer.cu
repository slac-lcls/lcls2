#include "LcReducer.hh"

#include "GpuAsyncLib.hh"


using namespace Drp::Gpu;


LcReducer::LcReducer(Parameters& para, MemPoolGpu& pool) :
  Gpu::ReducerAlgo(&para, &pool)
{
}

// This routine records the graph that does the data reduction
void LcReducer::recordGraph(hipStream_t&             stream,
                            const unsigned            index,
                            float* const __restrict__ calibBuffers,
                            float* const __restrict__ dataBuffers)
{
  int* d_fullcarry;
  hipMalloc((void **)&d_fullcarry, chunks * sizeof(int));
  d_reset<<<1, 1>>>();
  hipMemset(d_fullcarry, 0, chunks * sizeof(byte));
  d_encode<<<blocks, TPB>>>(dpreencdata, dpreencsize, d_encoded, d_encsize, d_fullcarry);
  hipFree(d_fullcarry);
  hipDeviceSynchronize();
}

// The class factory

extern "C" Drp::Gpu::ReducerAlgo* createReducer(Drp::Parameters& para, Drp::Gpu::MemPoolGpu& pool)
{
  return new Drp::Gpu::LcReducer(para, pool);
}
