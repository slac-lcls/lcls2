#include "LcReducer.hh"

#include "GpuAsyncLib.hh"


using namespace Drp::Gpu;


LcReducer::LcReducer(const Parameters& para, const MemPoolGpu& pool) :
  Gpu::ReducerAlgo(para, pool, Alg("LC", 0, 0, 0))
{
}

// This routine records the graph that does the data reduction
void LcReducer::recordGraph(hipStream_t&      stream,
                            const unsigned&    index,
                            float const* const calibBuffers,
                            const size_t       calibBufsCnt,
                            uint8_t    * const dataBuffers,
                            const size_t       dataBufsCnt,
                            unsigned*          extent)
{
  int* d_fullcarry;
  hipMalloc((void **)&d_fullcarry, chunks * sizeof(int));
  d_reset<<<1, 1>>>();
  hipMemset(d_fullcarry, 0, chunks * sizeof(byte));
  d_encode<<<blocks, TPB>>>(dpreencdata, dpreencsize, d_encoded, d_encsize, d_fullcarry);
  hipFree(d_fullcarry);
  hipDeviceSynchronize();
}

// The class factory

extern "C" Drp::Gpu::ReducerAlgo* createReducer(const Drp::Parameters& para, Drp::Gpu::MemPoolGpu& pool)
{
  return new Drp::Gpu::LcReducer(para, pool);
}
