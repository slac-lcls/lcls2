#include "hip/hip_runtime.h"
#include "Collector.hh"

#include "Detector.hh"
#include "Reader.hh"

#include "psalg/utils/SysLog.hh"
#include "psdaq/service/EbDgram.hh"     // For TimingHeader
#include "psdaq/trigger/TriggerPrimitive.hh"
#include "psdaq/eb/eb.hh"

using namespace XtcData;
using namespace Pds;
using namespace Drp;
using namespace Drp::Gpu;
using logging = psalg::SysLog;
using us_t = std::chrono::microseconds;

static const char* const RED_ON  = "\033[0;31m";
static const char* const RED_OFF = "\033[0m";
static const unsigned EvtCtrMask = 0xffffff;


Collector::Collector(const Parameters&         para,
                     MemPoolGpu&               pool,
                     std::vector<Reader>&      readers,
                     Trg::TriggerPrimitive*    triggerPrimitive,
                     const std::atomic<bool>&  terminate_h,
                     const cuda::atomic<int>&  terminate_d) :
  m_pool            (pool),
  m_triggerPrimitive(triggerPrimitive),
  m_terminate_h     (terminate_h),
  m_terminate_d     (terminate_d),
  m_graph           (0),
  m_last            (0),
  m_lastPid         (0),
  m_latPid          (0),
  m_lastComplete    (0),
  m_lastTid         (TransitionId::Unconfigure),
  m_para            (para)
{
  // Gather buffer index queues for Reader to Collector comms
  chkError(hipMalloc(&m_readerQueues_d, readers.size() * sizeof(*m_readerQueues_d)));
  for (unsigned i = 0; i < readers.size(); ++i) {
    chkError(hipMemcpy(&m_readerQueues_d[i], readers[i].queue(), sizeof(*m_readerQueues_d), hipMemcpyHostToDevice));
  }

  // Set up buffer index queue for Collector to Host comms
  m_collectorQueue.h = new Gpu::RingIndexDtoH(pool.nbuffers(), m_terminate_h, m_terminate_d);
  chkError(hipMalloc(&m_collectorQueue.d,                     sizeof(*m_collectorQueue.d)));
  chkError(hipMemcpy( m_collectorQueue.d, m_collectorQueue.h, sizeof(*m_collectorQueue.d), hipMemcpyHostToDevice));

  /** Create the Collector EB stream **/
  chkFatal(hipStreamCreateWithFlags(&m_stream, hipStreamNonBlocking));
  logging::debug("Done with creating collector stream");

  // Set up a done flag to cache m_terminate's value and avoid some PCIe transactions
  chkError(hipMalloc(&m_done,    sizeof(*m_done)));
  chkError(hipMemset( m_done, 0, sizeof(*m_done)));

  // Keep track of the head and tail indices of the Collector stream
  chkError(hipMalloc(&m_head,    sizeof(*m_head)));
  chkError(hipMemset( m_head, 0, sizeof(*m_head)));
  chkError(hipMalloc(&m_tail,    sizeof(*m_tail)));
  chkError(hipMemset( m_tail, 0, sizeof(*m_tail)));

  // Prepare the Collector graph
  if (_setupGraph()) {
    logging::critical("Failed to set up Collector graph");
    abort();
  }
}

Collector::~Collector()
{
  printf("*** Collector dtor 1\n");
  chkError(hipGraphExecDestroy(m_graphExec));
  chkError(hipGraphDestroy(m_graph)); // @todo: Goes away?
  printf("*** Collector dtor 2\n");

  chkError(hipFree(m_tail));
  printf("*** Collector dtor 2a\n");
  chkError(hipFree(m_head));
  printf("*** Collector dtor 2b\n");
  chkError(hipFree(m_done));
  printf("*** Collector dtor 3\n");

  chkError(hipStreamDestroy(m_stream));
  printf("*** Collector dtor 4\n");

  chkError(hipFree(m_collectorQueue.d));
  delete m_collectorQueue.h;
  printf("*** Collector dtor 5\n");

  chkError(hipFree(m_readerQueues_d));
  printf("*** Collector dtor 6\n");
}

int Collector::_setupGraph()
{
  printf("*** Collector setupGraph 1\n");
  // Build the graph
  if (m_graph == 0) {        // @todo: Graphs can be created on the stack
    logging::debug("Recording collector graph");
    auto& hostWriteBufs = m_pool.hostWrtBufsVec_h();
    for (unsigned panel = 0; panel < m_pool.panels().size(); ++panel) {
      printf("*** Collector setupGraph attach 1: panel %u, sz %zu\n", panel, m_collectorQueue.h->size());
      // hostWriteBufs[panel][0] is the base pointer for the entire allocation, i.e., all nBuffers
      chkError(hipStreamAttachMemAsync(m_stream, hostWriteBufs[panel][0], 0, hipMemAttachHost));
      printf("*** Collector setupGraph attach 2\n");
    }
    printf("*** Collector setupGraph 2\n");
    m_graph = _recordGraph(m_stream);
    if (m_graph == 0)
      return -1;
  }
  printf("*** Collector setupGraph 3\n");

  // Instantiate the graph
  if (chkError(hipGraphInstantiate(&m_graphExec, m_graph, hipGraphInstantiateFlagDeviceLaunch),
               "Collector graph create failed")) {
    return -1;
  }
  printf("*** Collector setupGraph 4\n");

  // @todo: No need to hang on to the stream info
  //hipGraphDestroy(m_graph);

  // Upload the graph so it can be launched by the scheduler kernel later
  logging::debug("Uploading Collector graph...");
  if (chkError(hipGraphUpload(m_graphExec, m_stream), "Collector graph upload failed")) {
    return -1;
  }
  printf("*** Collector setupGraph 5\n");

  return 0;
}

// This kernel collects and event builds contributions from the DMA streams
static __global__ void _collector(unsigned*         __restrict__ head,
                                  unsigned*         __restrict__ tail,
                                  RingIndexDtoD*    __restrict__ readerQueues,
                                  RingIndexDtoH&                 collectorQueue,
                                  uint32_t** const* __restrict__ in,
                                  const cuda::atomic<int>&       terminate,
                                  bool*             __restrict__ done)
{
  printf("*** _collector 1 tail %u, head %u\n", *tail, *head);
  int panel = blockIdx.x * blockDim.x + threadIdx.x;

  // Refresh the head if the tail has caught up to it
  // It might be desireable to refresh the head on every call, but that could
  // prevent progressing the tail toward the head since it blocks when there
  // is no change.  @todo: Revisit this
  if (*tail == *head) {
    printf("*** _collector 2\n");
    __shared__ unsigned hd0;

    // Get one intermediate buffer index per FPGA
    unsigned hdN;
    while ((hdN = readerQueues[panel].consume()) == *head) { // This can block
      if ( (*done = terminate.load(cuda::memory_order_acquire)) )  return;
    }
    printf("*** _collector 3, hdN %u\n", hdN);
    if (panel == 0)  hd0 = hdN;

    printf("*** _collector 4, hd0 %u\n", hd0);
    // @todo: grp.sync();
    __syncthreads();
    printf("*** _collector 5\n");

    if (hdN != hd0) {                   // Do this even for panel == 0?
      printf("Index mismatch for FPGA[%u]: %u != %u", panel, hdN, hd0);
      return;                           // abort(); ???
    }
    // Advance head
    if (panel == 0) *head = hdN;
  }

  printf("*** _collector 6, tail %u, head %u\n", *tail, *head);
  printf("*** _collector 6a, in %p\n", in);
  printf("*** _collector 6b, in[%u] %p\n", panel, in[panel]);
  printf("*** _collector 6c, in[%u][%u] %p\n", panel, *tail, in[panel][*tail]);
  printf("*** _collector 6d, in[%u][%u][0] %08x\n", panel, *tail, in[panel][*tail][0]);
  printf("*** _collector 6e, in[%u][%u][1] %08x\n", panel, *tail, in[panel][*tail][1]);
  printf("*** _collector 6f, in[%u][%u][8] %08x\n", panel, *tail, in[panel][*tail][8]);
  printf("*** _collector 6g, in[%u][%u][9] %08x\n", panel, *tail, in[panel][*tail][9]);

  // Check that the Pulse ID is the same for all FPGAs
  const unsigned  thOs = sizeof(DmaDsc) / sizeof(***in);
  const uint64_t& pid0 = *(uint64_t*)(&in[    0][*tail][thOs]);
  const uint64_t& pidN = *(uint64_t*)(&in[panel][*tail][thOs]);
  if (pidN != pid0) {
    // @todo: These should be counted these instead of printed...
    printf("Pulse ID mismatch for FPGA[%u] @ index %u: %014lx != %014lx", panel, *tail, pidN, pid0);
    return;                             // abort(); ???
  }
  printf("*** _collector 7, pid %014lx, env %08x\n", pid0, in[0][*tail][thOs+4]);

  // @todo: Copy only one device's DmaDsc and TimingHeader to the host?
  //        Currently, these are in managed memory, which resides on the device
  //        but with addresses the host can access.  Perhaps the transfer over
  //        PCIe is done only when the host does such an access, in which case
  //        there would seem to be no benefit to keeping these structures in
  //        non-managed device memory and then memcpying one of them to the host.
}

// This will re-launch the current graph
static __global__ void _graphLoop(unsigned*      idx,
                                  RingIndexDtoH& collectorQueue,
                                  const bool&    done)
{
  printf("*** Collector graphLoop 1\n");
  if (done)  return;
  printf("*** Collector graphLoop 1a, idx %u\n", *idx);

  // Push index to host
  *idx = collectorQueue.produce(*idx);
  printf("*** Collector graphLoop 2, idx %u\n", *idx);

  hipGraphLaunch(cudaGetCurrentGraphExec(), cudaStreamGraphTailLaunch);
  printf("*** Collector graphLoop 3\n");
}

hipGraph_t Collector::_recordGraph(hipStream_t& stream)
{
  printf("*** Collector::record 1\n");
  auto hostWrtBufs_d = m_pool.hostWrtBufs_d();
  auto calibBuffers  = m_pool.calibBuffers_d();

  printf("*** Collector::record 2\n");
  if (chkError(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal),
               "Collector stream begin capture failed")) {
    return 0;
  }
  printf("*** Collector::record 3, head %p, tail %p\n", m_head, m_tail);

  // Collect and event build data from the PGP FPGAs
  _collector<<<1, m_pool.panels().size(), 1, stream>>>(m_head,
                                                       m_tail,
                                                       m_readerQueues_d,
                                                       *m_collectorQueue.d,
                                                       hostWrtBufs_d,
                                                       m_terminate_d,
                                                       m_done);
  printf("*** Collector::record 4, trgPrmtv %p\n", m_triggerPrimitive);

  // Process calibBuffers[tail] into TEB input data placed at the end of hostWriteBufs[tail]
  // @todo: Deal with transitions
  // @todo: Provide a GPU-enabled base class for TriggerPrimitive
  if (m_triggerPrimitive) { // else this DRP doesn't provide TEB input
    m_triggerPrimitive->event(stream,
                              calibBuffers,
                              hostWrtBufs_d,
                              *m_tail,
                              *m_done);
  }
  printf("*** Collector::record 5\n");

  // Re-launch! Additional behavior can be put in graphLoop as needed. For now, it just re-launches the current graph.
  _graphLoop<<<1, 1, 0, stream>>>(m_tail, *m_collectorQueue.d, *m_done);
  printf("*** Collector::record 6\n");

  hipGraph_t graph;
  if (chkError(hipStreamEndCapture(stream, &graph),
               "Collector stream end capture failed")) {
    return 0;
  }

  return graph;
}

void Collector::start()
{
  logging::info("Collector starting");
  chkError(hipCtxSetCurrent(m_pool.context().context()));  // Needed, else kernels misbehave

  resetEventCounter();

  // Launch the Collector graph
  chkFatal(hipGraphLaunch(m_graphExec, m_stream));
}

void Collector::freeDma(unsigned index)
{
  m_collectorQueue.h->release(index);
  // @todo: Make the host version accessible
  //for (unsigned i = 0; i < m_pool.panels().size(); ++i) {
  //  m_readerQueues_h[i]->release(index);
  //}

  m_pool.freeDma(1, nullptr);
}

void Collector::freeDma(PGPEvent* event)
{
  const uint32_t lane = 0;                   // The lane is always 0 for GPU-enabled PGP devices
  DmaBuffer* buffer = &event->buffers[lane];
  event->mask = 0;
  freeDma(buffer->index);
}

unsigned Collector::_checkDmaDsc(unsigned index) const
{
  unsigned rc = 0;
  const auto dsc0 = (DmaDsc*)(m_pool.hostWrtBufsVec_h()[0][index]);

  logging::debug("panel %d: dma %d hdr: err %08x,  sz %08x, rsvd %08x %08x %08x %08x %08x %08x",
                 0, index, dsc0->error, dsc0->size, dsc0->_rsvd[0], dsc0->_rsvd[1], dsc0->_rsvd[2],
                 dsc0->_rsvd[3], dsc0->_rsvd[4], dsc0->_rsvd[5]);

  for (unsigned i = 1; i < m_pool.panels().size(); ++i) {
    bool ne = false;
    const auto dscN = (DmaDsc*)(m_pool.hostWrtBufsVec_h()[i][index]);
    ne |= dscN->error != dsc0->error;
    ne |= dscN->size  != dsc0->size;

    if (ne) {
      logging::debug("panel %d: idx %d dma: err %08x,  sz %08x, rsvd %08x %08x %08x %08x %08x %08x",
                     i, index, dscN->error, dscN->size, dscN->_rsvd[0], dscN->_rsvd[1], dscN->_rsvd[2],
                     dscN->_rsvd[3], dscN->_rsvd[4], dscN->_rsvd[5]);
      rc |= 1;                          // If different, include panel 0 in the list
    }
    rc |= 1 << i;
  }

  return rc;
}

unsigned Collector::_checkTimingHeader(unsigned index) const
{
  unsigned rc = 0;
  const auto dsc0 = (DmaDsc*)(m_pool.hostWrtBufsVec_h()[0][index]);
  const auto th0  = (TimingHeader*)&dsc0[1];

  logging::debug("panel %d: idx %d  th: ctl %02x, pid %014lx, ts %016lx, env %08x, ctr %08x, opq %08x %08x",
                 0, index, th0->control(), th0->pulseId(), th0->time.value(), th0->env, th0->evtCounter,
                 th0->_opaque[0], th0->_opaque[1]);

  for (unsigned i = 1; i < m_pool.panels().size(); ++i) {
    bool ne = false;
    const auto dscN = (DmaDsc*)(m_pool.hostWrtBufsVec_h()[i][index]);
    const auto thN  = (TimingHeader*)&dscN[1];
    ne |= thN->control()    != th0->control();
    ne |= thN->pulseId()    != th0->pulseId();
    ne |= thN->time.value() != th0->time.value();
    ne |= thN->env          != th0->env;
    ne |= thN->evtCounter   != th0->evtCounter;

    if (ne) {
      logging::debug("panel %d: dma %d  th: ctl %02x, pid %014lx, ts %u.%09u, env %08x, ctr %08x, opq %08x %08x",
                     i, index, thN->control(), thN->pulseId(), thN->time.seconds(), thN->time.nanoseconds(),
                     thN->env, thN->evtCounter, thN->_opaque[0], thN->_opaque[1]);
      rc |= 1;                          // If different, include panel 0 in the list
    }
    rc |= 1 << i;
  }

  return rc;
}

unsigned Collector::receive(Detector* det, CollectorMetrics& metrics)
{
  const auto& hostWriteBufs = m_pool.hostWrtBufsVec_h()[0]; // When no error, hdrs in all are the same
  const uint32_t bufferMask = m_collectorQueue.h->size() - 1;

  unsigned head = m_collectorQueue.h->consume(); // This can block
  unsigned tail = m_last;
  while (tail != head) {
    const volatile auto dsc = (DmaDsc*)(hostWriteBufs[tail]);
    const volatile auto th  = (TimingHeader*)&dsc[1];

    uint64_t pid;
    while (!m_terminate_h.load(std::memory_order_acquire)) {
      pid = th->pulseId();
      if (pid > m_lastPid)  break;
      if (!m_lastPid && !pid)  break; // Expect lastPid to be 0 only on startup
    }
    if (m_terminate_h.load(std::memory_order_acquire))  break;
    if (!pid)  continue;              // Search for a DMA buffer with data in it
    m_lastPid = pid;

    // Handle the case when the headers don't match across panels
    // @todo: Too expensive?  This fetches the headers of all panels from the GPU
    //        Maybe do the test on the GPU and set a flag if they differ and
    //        print here when it is set
    unsigned dmas, ths;
    if ( (dmas = _checkDmaDsc(tail)) || (ths = _checkTimingHeader(tail)) ) {
      // Assume we can recover from non-matching panel headers
      logging::error("Headers differ between panels: DmaDsc: %02x, TimingHeader: %02x", dmas, ths);
      freeDma(tail);                    // Leaves event mask = 0
      metrics.m_nHdrMismatch += 1;
      continue;
    }

    uint32_t size = dsc->size;          // Size of the DMA
    uint32_t index = tail;
    uint32_t lane = 0;      // The lane is always 0 for GPU-enabled PGP devices
    metrics.m_dmaSize   = size;
    metrics.m_dmaBytes += size;

    // Check for DMA buffer overflow
    if (dsc->error & 0x4) {
      logging::critical("%d DMA overflowed buffer: %d vs %d", tail, size, m_pool.dmaSize());
      abort();                          // @todo: Still necessary to abort?
    }

    const Pds::TimingHeader* timingHeader = det->getTimingHeader(tail);

    // Measure TimingHeader arrival latency as early as possible
    if (timingHeader->pulseId() - m_latPid > 1300000/14) { // 10 Hz
        metrics.m_latency = Eb::latency<us_t>(timingHeader->time);
        m_latPid = timingHeader->pulseId();
    }
    if (timingHeader->error()) {
        if (metrics.m_nTmgHdrError < 5) { // Limit prints at rate
            logging::error("Timing header error bit is set");
        }
        metrics.m_nTmgHdrError += 1;
    }

    uint32_t evtCounter = timingHeader->evtCounter & EvtCtrMask;
    unsigned pgpIndex = evtCounter & bufferMask;
    PGPEvent*  event  = &m_pool.pgpEvents[pgpIndex];
    if (event->mask)  printf("*** PGPEvent mask != 0 for ctr %d\n", pgpIndex);
    DmaBuffer* buffer = &event->buffers[lane]; // @todo: Do we care about this?
    buffer->size = size;                       //   "
    buffer->index = index;                     //   "
    event->mask |= (1 << lane);

    m_pool.allocateDma(); // DMA buffer was allocated when f/w incremented evtCounter

    if (dsc->error) {
      // Assume we can recover from non-overflow DMA errors
      if (metrics.m_nDmaErrors < 5) {   // Limit prints at rate
        logging::error("DMA error 0x%x", dsc->error);
      }
      // This assumes the DMA succeeded well enough that evtCounter is valid
      handleBrokenEvent(*event);
      freeDma(event);                   // Leaves event mask = 0
      metrics.m_nDmaErrors += 1;
      continue;
    }

    XtcData::TransitionId::Value transitionId = timingHeader->service();
    const uint32_t* data = reinterpret_cast<const uint32_t*>(timingHeader);
    logging::debug("PGPReader  size %u  hdr %016lx.%016lx.%08x  err 0x%x",
                   size,
                   reinterpret_cast<const uint64_t*>(data)[0], // PulseId
                   reinterpret_cast<const uint64_t*>(data)[1], // Timestamp
                   reinterpret_cast<const uint32_t*>(data)[4], // env
                   dsc->error);

    if (transitionId == TransitionId::BeginRun) {
      resetEventCounter();              // Compensate for the ClearReadout sent before BeginRun
    }
    if (evtCounter != ((m_lastComplete + 1) & EvtCtrMask)) {
      if (m_lastTid != TransitionId::Unconfigure) {
        if ((metrics.m_nPgpJumps < 5) || m_para.verbose) { // Limit prints at rate
          auto evtCntDiff = evtCounter - m_lastComplete;
          logging::error("%sPGPReader: Jump in TimingHeader evtCounter %u -> %u | difference %d, DMA size %u%s",
                         RED_ON, m_lastComplete, evtCounter, evtCntDiff, size, RED_OFF);
          logging::error("new data: %08x %08x %08x %08x %08x %08x  (%s)",
                         data[0], data[1], data[2], data[3], data[4], data[5], TransitionId::name(transitionId));
          logging::error("lastData: %08x %08x %08x %08x %08x %08x  (%s)",
                         m_lastData[0], m_lastData[1], m_lastData[2], m_lastData[3], m_lastData[4], m_lastData[5], TransitionId::name(m_lastTid));
        }
        handleBrokenEvent(*event);
        freeDma(event);                 // Leaves event mask = 0
        metrics.m_nPgpJumps += 1;
        continue;                       // Throw away out-of-sequence events
      } else if (transitionId != TransitionId::Configure) {
        freeDma(event);                 // Leaves event mask = 0
        continue;                       // Drain
      }
    }
    m_lastComplete = evtCounter;
    m_lastTid = transitionId;
    memcpy(m_lastData, data, 24);

    auto rogs = timingHeader->readoutGroups();
    if ((rogs & (1 << m_para.partition)) == 0) {
      logging::debug("%s @ %u.%09u (%014lx) without common readout group (%u) in env 0x%08x",
                     XtcData::TransitionId::name(transitionId),
                     timingHeader->time.seconds(), timingHeader->time.nanoseconds(),
                     timingHeader->pulseId(), m_para.partition, timingHeader->env);
      ++m_lastComplete;
      handleBrokenEvent(*event);
      freeDma(event);                   // Leaves event mask = 0
      metrics.m_nNoComRoG += 1;
      continue;
    }
    if (transitionId == XtcData::TransitionId::SlowUpdate) {
      uint16_t missingRogs = m_para.rogMask & ~rogs;
      if (missingRogs) {
        logging::debug("%s @ %u.%09u (%014lx) missing readout group(s) (0x%04x) in env 0x%08x",
                       XtcData::TransitionId::name(transitionId),
                       timingHeader->time.seconds(), timingHeader->time.nanoseconds(),
                       timingHeader->pulseId(), missingRogs, timingHeader->env);
        ++m_lastComplete;
        handleBrokenEvent(*event);
        freeDma(event);                 // Leaves event mask = 0
        metrics.m_nMissingRoGs += 1;
        continue;
      }
    }

    if (transitionId != XtcData::TransitionId::L1Accept) {
      if (transitionId != XtcData::TransitionId::SlowUpdate) {
        logging::info("PGPReader  saw %s @ %u.%09u (%014lx)",
                      XtcData::TransitionId::name(transitionId),
                      timingHeader->time.seconds(), timingHeader->time.nanoseconds(),
                      timingHeader->pulseId());
      }
      else {
        logging::debug("PGPReader  saw %s @ %u.%09u (%014lx)",
                       XtcData::TransitionId::name(transitionId),
                       timingHeader->time.seconds(), timingHeader->time.nanoseconds(),
                       timingHeader->pulseId());
      }
    }

    metrics.m_nevents += 1;
    tail = (tail + 1) & bufferMask;
  }
  unsigned nEvents = (head - m_last) & bufferMask;
  m_last = tail;

  return nEvents;
}
