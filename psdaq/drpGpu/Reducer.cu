#include "hip/hip_runtime.h"
#include "Reducer.hh"

#include "Detector.hh"

#include "psalg/utils/SysLog.hh"
#include "psdaq/service/MetricExporter.hh"
#include "psdaq/eb/ResultDgram.hh"
#include "ReducerAlgo.hh"

#include <sys/prctl.h>

using logging = psalg::SysLog;
using namespace XtcData;
using namespace Drp;
using namespace Drp::Gpu;
using namespace Pds;
using namespace Pds::Eb;

using us_t = std::chrono::microseconds;


Reducer::Reducer(const Parameters&            para,
                 MemPoolGpu&                  pool,
                 Detector&                    det,
                 const std::atomic<bool>&     terminate,
                 const cuda::atomic<uint8_t>& terminate_d) :
  m_pool       (pool),
  m_terminate  (terminate),
  m_terminate_d(terminate_d),
  m_reduce_us  (0),
  m_para       (para)
{
  // Create the Reducer streams
  m_streams.resize(m_para.nworkers);
  m_t0.resize(m_para.nworkers);
  m_heads_h.resize(m_para.nworkers);
  m_heads_d.resize(m_para.nworkers);
  m_tails_h.resize(m_para.nworkers);
  m_tails_d.resize(m_para.nworkers);
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    chkFatal(hipStreamCreateWithFlags(&m_streams[i], hipStreamNonBlocking));

    // Keep track of the head and tail indices of the Reducer stream
    chkError(hipHostAlloc(&m_heads_h[i], sizeof(*m_heads_h[i]), hipHostMallocDefault));
    chkError(hipHostGetDevicePointer(&m_heads_d[i], m_heads_h[i], 0));
    *m_heads_h[i] = 0;
    chkError(hipHostAlloc(&m_tails_h[i], sizeof(*m_tails_h[i]), hipHostMallocDefault));
    chkError(hipHostGetDevicePointer(&m_tails_d[i], m_tails_h[i], 0));
    *m_tails_h[i] = 0;
  }
  logging::debug("Done with creating %u Reducer streams", m_streams.size());

  // Set up the reducer algorithm instances
  m_algos.resize(m_para.nworkers);
  if (!_setupAlgos(det)) {
    logging::critical("Error setting up Reducer Algorithm instances");
    abort();
  }

  // The header consists of the Dgram with the parent Xtc, the ShapesData Xtc, the
  // Shapes Xtc with its payload and Data Xtc, the payload of which is on the GPU.
  auto headerSize  = sizeof(Dgram) + 3 * sizeof(Xtc) + MaxRank * sizeof(uint32_t);
  auto payloadSize = m_algos[0]->payloadSize(); // Each instance returns the same value
  auto totalSize   = headerSize + payloadSize;
  if (totalSize < m_para.maxTrSize)  payloadSize = m_para.maxTrSize - headerSize;

  // Prepare buffers to receive the reduced data,
  // prepended with some reserved space for the datagram header.
  // The application sees only the pointer to the data buffer.
  m_pool.createReduceBuffers(payloadSize, headerSize);

  // Prepare the CUDA graphs
  if (true) { //m_algos[0]->hasGraph()) {         // Same value for all instances
    m_graphExecs.resize(m_streams.size());
    for (unsigned i = 0; i < m_para.nworkers; ++i) {
      if (_setupGraph(i)) {
        logging::critical("Failed to set up Reducer graph");
        abort();
      }
    }
  }

  // Set up the worker queues to fit all buffers without getting full
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    m_inputQueues.emplace_back(SPSCQueue<unsigned>(m_pool.nbuffers()));
    m_outputQueues.emplace_back(SPSCQueue<size_t>(m_pool.nbuffers()));
  }

  // Start the worker threads
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    m_threads.emplace_back(&Reducer::_worker,
                           std::ref(*this),
                           i,
                           std::ref(m_inputQueues[i]),
                           std::ref(m_outputQueues[i]));
  }
}

Reducer::~Reducer()
{
  if (m_threads.size())
    logging::info("Shutting down reducer workers");
  for (unsigned i = 0; i < m_threads.size(); i++) {
    m_inputQueues[i].shutdown();
    if (m_threads[i].joinable()) {
      m_threads[i].join();
    }
  }
  if (m_threads.size()) {
    logging::info("Reducer worker threads finished");
  }
  for (unsigned i = 0; i < m_outputQueues.size(); i++) {
    m_outputQueues[i].shutdown();
  }
  m_outputQueues.clear();
  m_inputQueues.clear();
  m_threads.clear();

  printf("*** Reducer dtor 1\n");
  for (auto& graphExec : m_graphExecs) {
    chkError(hipGraphExecDestroy(graphExec));
  }
  m_graphExecs.clear();
  printf("*** Reducer dtor 2\n");

  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    if (m_algos[i])  delete m_algos[i];
  }
  m_algos.clear();
  m_dl.close();
  printf("*** Reducer dtor 3\n");

  m_pool.destroyReduceBuffers();
  printf("*** Reducer dtor 4\n");

  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    chkError(hipHostFree(m_tails_h[i]));
    chkError(hipHostFree(m_heads_h[i]));

    chkError(hipStreamDestroy(m_streams[i]));
  }
  m_heads_h.clear();
  m_heads_d.clear();
  m_tails_h.clear();
  m_tails_d.clear();
  m_streams.clear();
  printf("*** Reducer dtor 5\n");
}

int Reducer::setupMetrics(const std::shared_ptr<MetricExporter> exporter,
                          std::map<std::string, std::string>&   labels)
{
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    auto wkr = std::to_string(i);
    exporter->add("DRP_inputQueue"+wkr,  labels, MetricType::Gauge, [&, i](){ return m_inputQueues[i].guess_size(); });
    exporter->add("DRP_outputQueue"+wkr, labels, MetricType::Gauge, [&, i](){ return m_outputQueues[i].guess_size(); });
  }

  exporter->add("DRP_reduceTime", labels, MetricType::Gauge, [&](){ return m_reduce_us; });

  return 0;
}

bool Reducer::_setupAlgos(Detector& det)
{
  // @todo: In the future, find out which Reducer to load from the Detector's configDb entry
  //        For now, load it according to a command line kwarg parameter
  std::string reducer;
  if (m_para.kwargs.find("reducer") == m_para.kwargs.end()) {
    logging::error("Missing required kwarg 'reducer'");
    return false;
  }
  reducer = const_cast<Parameters&>(m_para).kwargs["reducer"];

  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    if (m_algos[i])  delete m_algos[i]; // If the object exists, delete it
  }
  m_dl.close();                         // If a lib is open, close it first

  const std::string soName("lib"+reducer+".so");
  logging::debug("Loading library '%s'", soName.c_str());
  if (m_dl.open(soName, RTLD_LAZY)) {
    logging::error("Error opening library '%s'", soName.c_str());
    return false;
  }
  const std::string symName("createReducer");
  auto createFn = m_dl.loadSymbol(symName.c_str());
  if (!createFn) {
    logging::error("Symbol '%s' not found in %s",
                   symName.c_str(), soName.c_str());
    return false;
  }
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    auto instance = reinterpret_cast<reducerAlgoFactoryFn_t*>(createFn)(m_para, m_pool, det);
    if (!instance)
    {
      logging::error("Error calling %s from %s", symName.c_str(), soName.c_str());
      return false;
    }
    m_algos[i] = instance;
  }
  return true;
}

int Reducer::_setupGraph(unsigned instance)
{
  hipGraph_t      graph;
  hipGraphExec_t& graphExec = m_graphExecs[instance];
  hipStream_t     stream    = m_streams[instance];

  // Build the graph
  logging::debug("Recording Reducer graph %u", instance);
  graph = _recordGraph(instance);
  if (graph == 0) {
    return -1;
  }

  // Instantiate the executable graph
  if (chkError(hipGraphInstantiate(&graphExec, graph, hipGraphInstantiateFlagDeviceLaunch),
               "Reducer graph create failed")) {
    return -1;
  }

  // No need to hang on to the stream info
  hipGraphDestroy(graph);

  // Upload the graph so it can be launched by the scheduler kernel later
  logging::debug("Uploading Reducer graph %u...", instance);
  if (chkError(hipGraphUpload(graphExec, stream), "Reducer graph upload failed")) {
    return -1;
  }

  return 0;
}

/** This kernel receives a message from TebReceiver that indicates which
 * calibBuffer is ready for reducing.
 */
static __global__ void _receive(unsigned* const __restrict__ head,
                                unsigned* const __restrict__ tail,
                                const cuda::atomic<uint8_t>& terminate)
{
  //printf("### _receive 1 done %d, tail %u, head %u\n", terminate.load(), *tail, *head);

  // Wait for the head to advance with respect to the tail
  auto t = *tail;
  while (*head == t) {
    if (terminate.load(cuda::memory_order_acquire))  break;
  }
  //printf("### Reducer receive:   h %u, t %u, d %d\n", *head, t, terminate.load());
}

/** This will re-launch the current graph */
static __global__ void _graphLoop(unsigned* const __restrict__ head,
                                  unsigned* const __restrict__ tail,
                                  const cuda::atomic<uint8_t>& terminate)
{
  //printf("### Reducer graphLoop: 1, done %d, idx %u\n", terminate.load(), *index);
  if (terminate.load(cuda::memory_order_acquire))  return;

  //printf("### Reducer graphLoop: 2 t %u, h %u\n", *tail, *head);

  // Signal that this worker is done
  *tail = *head;                   // With nworkers > 1, head - tail may be > 1

  // Commented out to let TebRcvr::complete() launch the graph
  //hipGraphLaunch(cudaGetCurrentGraphExec(), cudaStreamGraphTailLaunch);
  //printf("### Reducer graphLoop: 3\n");
}

hipGraph_t Reducer::_recordGraph(unsigned instance)
{
  auto stream       = m_streams[instance];
  auto calibBuffers = m_pool.calibBuffers_d();
  auto calibBufsSz  = m_pool.calibBufsSize();
  auto calibBufsCnt = calibBufsSz / sizeof(*calibBuffers);
  auto dataBuffers  = m_pool.reduceBuffers_d();
  auto dataBufsRsvd = m_pool.reduceBufsReserved();
  auto dataBufsSz   = m_pool.reduceBufsSize();
  auto dataBufsCnt  = (dataBufsRsvd + dataBufsSz) / sizeof(*dataBuffers);

  if (chkError(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal),
               "Reducer stream begin capture failed")) {
    return 0;
  }

  // Handle messages from TebReceiver to process an event
  _receive<<<1, 1, 0, stream>>>(m_heads_d[instance], m_tails_d[instance], m_terminate_d);

  // Perform the reduction algorithm
  m_algos[instance]->recordGraph(stream,
                                 *m_heads_d[instance],
                                 calibBuffers,
                                 calibBufsCnt,
                                 dataBuffers,
                                 dataBufsCnt);

  // Re-launch! Additional behavior can be put in graphLoop as needed.
  _graphLoop<<<1, 1, 0, stream>>>(m_heads_d[instance],
                                  m_tails_d[instance],
                                  m_terminate_d);

  // Signal to the host that the worker is done
  //chkError(hipEventRecord(event, stream));

  hipGraph_t graph;
  if (chkError(hipStreamEndCapture(stream, &graph),
               "Reducer stream end capture failed")) {
    return 0;
  }

  return graph;
}

void Reducer::startup()
{
  // Launch the Reducer graphs
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    chkFatal(hipGraphLaunch(m_graphExecs[i], m_streams[i]));
  }
}

void Reducer::_worker(unsigned instance, SPSCQueue<unsigned>& inputQueue, SPSCQueue<size_t>& outputQueue)
{
  logging::info("Reducer worker %u is starting with process ID %lu", instance, syscall(SYS_gettid));
  char nameBuf[16];
  snprintf(nameBuf, sizeof(nameBuf), "ReducerWkr%d", instance);
  if (prctl(PR_SET_NAME, nameBuf, 0, 0, 0) == -1) {
    perror("prctl");
  }

  auto  algo   = m_algos[instance];
  auto  head   = m_heads_h[instance];
  auto  tail   = m_tails_h[instance];
  auto  stream = m_streams[instance];
  auto& graph  = m_graphExecs[instance];

  unsigned index;
  while (inputQueue.pop(index)) {
    if  (algo->hasGraph()) {
      // Wait for the graph to finish executing before updating head
      unsigned hd, tl;
      do {
        chkError(hipMemcpyAsync((void*)&hd, head, sizeof(*head), hipMemcpyDeviceToHost, stream));
        chkError(hipMemcpyAsync((void*)&tl, tail, sizeof(*tail), hipMemcpyDeviceToHost, stream));
        chkError(hipStreamSynchronize(stream));
        //printf("*** Reducer::start[%u]: tail %d, head %d\n", instance, tl, hd);
      } while (hd != tl);                     // Wait if the kernel is still processing
      chkError(hipMemcpyAsync((void*)head, &index, sizeof(index), hipMemcpyHostToDevice, stream));
      *head = index;
    }

    auto t0{fast_monotonic_clock::now(CLOCK_MONOTONIC)};

    // Launch the Reducer
    size_t dataSize;
    algo->reduce(graph, stream, index, &dataSize);

    // Wait for the graph to complete
    chkError(hipStreamSynchronize(stream));

    auto now{fast_monotonic_clock::now(CLOCK_MONOTONIC)};
    m_reduce_us = std::chrono::duration_cast<us_t>(now - t0).count();

    // Signal completion to the recorder
    outputQueue.push(dataSize);
  }

  logging::info("Reducer worker %u is exiting", instance);
}
