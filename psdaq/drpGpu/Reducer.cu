#include "hip/hip_runtime.h"
#include "Reducer.hh"

#include "Detector.hh"

#include "psalg/utils/SysLog.hh"
#include "psdaq/eb/ResultDgram.hh"
#include "ReducerAlgo.hh"

using logging = psalg::SysLog;
using namespace XtcData;
using namespace Drp;
using namespace Drp::Gpu;
using namespace Pds;
using namespace Pds::Eb;

using us_t = std::chrono::microseconds;


Reducer::Reducer(const Parameters&            para,
                 MemPoolGpu&                  pool,
                 Detector&                    det,
                 const std::atomic<bool>&     terminate,
                 const cuda::atomic<uint8_t>& terminate_d) :
  m_pool       (pool),
  m_algo       (nullptr),
  m_terminate  (terminate),
  m_terminate_d(terminate_d),
  m_reduce_us  (0),
  m_para       (para)
{
  // Create the Reducer streams
  m_streams.resize(m_para.nworkers);
  m_t0.resize(m_para.nworkers);
  m_heads_h.resize(m_para.nworkers);
  m_heads_d.resize(m_para.nworkers);
  m_tails_h.resize(m_para.nworkers);
  m_tails_d.resize(m_para.nworkers);
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    chkFatal(hipStreamCreateWithFlags(&m_streams[i], hipStreamNonBlocking));

    // Keep track of the head and tail indices of the Reducer stream
    chkError(hipHostAlloc(&m_heads_h[i], sizeof(*m_heads_h[i]), hipHostMallocDefault));
    chkError(hipHostGetDevicePointer(&m_heads_d[i], m_heads_h[i], 0));
    *m_heads_h[i] = 0;
    chkError(hipHostAlloc(&m_tails_h[i], sizeof(*m_tails_h[i]), hipHostMallocDefault));
    chkError(hipHostGetDevicePointer(&m_tails_d[i], m_tails_h[i], 0));
    *m_tails_h[i] = 0;
  }
  logging::debug("Done with creating %u Reducer streams", m_streams.size());

  // Set up the reducer algorithm
  m_algo = _setupAlgo(det);
  if (!m_algo) {
    logging::critical("Error setting up Reducer Algorithm");
    abort();
  }

  // The header consists of the Dgram with the parent Xtc, the ShapesData Xtc, the
  // Shapes Xtc with its payload and Data Xtc, the payload of which is on the GPU.
  auto headerSize  = sizeof(Dgram) + 3 * sizeof(Xtc) + MaxRank * sizeof(uint32_t);
  auto payloadSize = m_algo->payloadSize();
  auto totalSize   = headerSize + payloadSize;
  if (totalSize < m_para.maxTrSize)  payloadSize = m_para.maxTrSize - headerSize;

  // Prepare buffers to receive the reduced data,
  // prepended with some reserved space for the datagram header.
  // The application sees only the pointer to the data buffer.
  m_pool.createReduceBuffers(payloadSize, headerSize);

  // Prepare the CUDA graphs
  m_graphExecs.resize(m_streams.size());
  for (unsigned i = 0; i < m_streams.size(); ++i) {
    if (_setupGraph(i)) {
      logging::critical("Failed to set up Reducer graph");
      abort();
    }
  }
}

Reducer::~Reducer()
{
  printf("*** Reducer dtor 1\n");
  for (auto& graphExec : m_graphExecs) {
    chkError(hipGraphExecDestroy(graphExec));
  }
  printf("*** Reducer dtor 2\n");

  if (m_algo)  delete m_algo;
  m_dl.close();
  printf("*** Reducer dtor 3\n");

  m_pool.destroyReduceBuffers();
  printf("*** Reducer dtor 4\n");

  for (unsigned i = 0; i < m_streams.size(); ++i) {
    chkError(hipHostFree(m_tails_h[i]));
    chkError(hipHostFree(m_heads_h[i]));

    chkError(hipStreamDestroy(m_streams[i]));
  }
  printf("*** Reducer dtor 5\n");
}

ReducerAlgo* Reducer::_setupAlgo(Detector& det)
{
  // @todo: In the future, find out which Reducer to load from the Detector's configDb entry
  //        For now, load it according to a command line kwarg parameter
  std::string reducer;
  if (m_para.kwargs.find("reducer") == m_para.kwargs.end()) {
    logging::error("Missing required kwarg 'reducer'");
    return nullptr;
  }
  reducer = const_cast<Parameters&>(m_para).kwargs["reducer"];

  if (m_algo)  delete m_algo;     // If the object exists, delete it
  m_dl.close();                   // If a lib is open, close it first

  const std::string soName("lib"+reducer+".so");
  logging::debug("Loading library '%s'", soName.c_str());
  if (m_dl.open(soName, RTLD_LAZY)) {
    logging::error("Error opening library '%s'", soName.c_str());
    return nullptr;
  }
  const std::string symName("createReducer");
  auto createFn = m_dl.loadSymbol(symName.c_str());
  if (!createFn) {
    logging::error("Symbol '%s' not found in %s",
                   symName.c_str(), soName.c_str());
    return nullptr;
  }
  auto instance = reinterpret_cast<reducerAlgoFactoryFn_t*>(createFn)(m_para, m_pool, det);
  if (!instance)
  {
    logging::error("Error calling %s from %s", symName.c_str(), soName.c_str());
    return nullptr;
  }
  return instance;
}

int Reducer::_setupGraph(unsigned instance)
{
  hipGraph_t      graph;
  hipGraphExec_t& graphExec = m_graphExecs[instance];
  hipStream_t     stream    = m_streams[instance];

  // Build the graph
  logging::debug("Recording Reducer graph %u", instance);
  graph = _recordGraph(instance);
  if (graph == 0) {
    return -1;
  }

  // Instantiate the executable graph
  if (chkError(hipGraphInstantiate(&graphExec, graph, hipGraphInstantiateFlagDeviceLaunch),
               "Reducer graph create failed")) {
    return -1;
  }

  // No need to hang on to the stream info
  hipGraphDestroy(graph);

  // Upload the graph so it can be launched by the scheduler kernel later
  logging::debug("Uploading Reducer graph %u...", instance);
  if (chkError(hipGraphUpload(graphExec, stream), "Reducer graph upload failed")) {
    return -1;
  }

  return 0;
}

/** This kernel receives a message from TebReceiver that indicates which
 * calibBuffer is ready for reducing.
 */
static __global__ void _receive(unsigned* const __restrict__ head,
                                unsigned* const __restrict__ tail,
                                const cuda::atomic<uint8_t>& terminate)
{
  //printf("### _receive 1 done %d, tail %u, head %u\n", terminate.load(), *tail, *head);

  // Wait for the head to advance with respect to the tail
  auto t = *tail;
  while (*head == t) {
    if (terminate.load(cuda::memory_order_acquire))  break;
  }
  //printf("### Reducer receive:   h %u, t %u, d %d\n", *head, t, terminate.load());
}

/** This will re-launch the current graph */
static __global__ void _graphLoop(unsigned* const __restrict__ head,
                                  unsigned* const __restrict__ tail,
                                  const cuda::atomic<uint8_t>& terminate)
{
  //printf("### Reducer graphLoop: 1, done %d, idx %u\n", terminate.load(), *index);
  if (terminate.load(cuda::memory_order_acquire))  return;

  //printf("### Reducer graphLoop: 2 t %u, h %u\n", *tail, *head);

  // Signal that this worker is done
  *tail = *head;                   // With nworkers > 1, head - tail may be > 1

  // Commented out to let TebRcvr::complete() launch the graph
  //hipGraphLaunch(cudaGetCurrentGraphExec(), cudaStreamGraphTailLaunch);
  //printf("### Reducer graphLoop: 3\n");
}

hipGraph_t Reducer::_recordGraph(unsigned instance)
{
  auto stream       = m_streams[instance];
  auto calibBuffers = m_pool.calibBuffers_d();
  auto calibBufsSz  = m_pool.calibBufsSize();
  auto calibBufsCnt = calibBufsSz / sizeof(*calibBuffers);
  auto dataBuffers  = m_pool.reduceBuffers_d();
  auto dataBufsRsvd = m_pool.reduceBufsReserved();
  auto dataBufsSz   = m_pool.reduceBufsSize();
  auto dataBufsCnt  = (dataBufsRsvd + dataBufsSz) / sizeof(*dataBuffers);

  if (chkError(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal),
               "Reducer stream begin capture failed")) {
    return 0;
  }

  // Handle messages from TebReceiver to process an event
  _receive<<<1, 1, 0, stream>>>(m_heads_d[instance], m_tails_d[instance], m_terminate_d);

  // Perform the reduction algorithm
  m_algo->recordGraph(stream,
                      *m_heads_d[instance],
                      calibBuffers,
                      calibBufsCnt,
                      dataBuffers,
                      dataBufsCnt);

  // Re-launch! Additional behavior can be put in graphLoop as needed.
  _graphLoop<<<1, 1, 0, stream>>>(m_heads_d[instance],
                                  m_tails_d[instance],
                                  m_terminate_d);

  // Signal to the host that the worker is done
  //chkError(hipEventRecord(event, stream));

  hipGraph_t graph;
  if (chkError(hipStreamEndCapture(stream, &graph),
               "Reducer stream end capture failed")) {
    return 0;
  }

  return graph;
}

void Reducer::startup()
{
  // Launch the Reducer graphs
  for (unsigned i = 0; i < m_para.nworkers; ++i) {
    chkFatal(hipGraphLaunch(m_graphExecs[i], m_streams[i]));
  }
}

void Reducer::start(unsigned worker, unsigned index)
{
  auto  instance  = worker % m_para.nworkers;
  auto  head      = m_heads_h[instance];
  auto  stream    = m_streams[instance];
  auto& graphExec = m_graphExecs[instance];

  // Wait for the graph to finish executing before updating head
  //unsigned h, t;
  //do {
  //  chkError(hipMemcpyAsync((void*)&h, head, sizeof(*head), hipMemcpyDeviceToHost, stream));
  //  chkError(hipMemcpyAsync((void*)&t, tail, sizeof(*tail), hipMemcpyDeviceToHost, stream));
  //  chkError(hipStreamSynchronize(stream));
  //  //printf("*** Reducer::start[%u]: tail %d, head %d\n", instance, t, h);
  //} while (h != t);                     // Wait if the kernel is still processing
  //chkError(hipMemcpyAsync((void*)head, &index, sizeof(index), hipMemcpyHostToDevice, stream));
  *head = index;

  m_t0[instance] = fast_monotonic_clock::now(CLOCK_MONOTONIC);

  // Launch the Reducer graph
  chkFatal(hipGraphLaunch(graphExec, stream));
}

void Reducer::receive(unsigned worker, unsigned index)
{
  auto  instance = worker % m_para.nworkers;
  auto& stream   = m_streams[instance];

  // Wait for the graph to complete
  chkError(hipStreamSynchronize(stream));

  auto now{fast_monotonic_clock::now(CLOCK_MONOTONIC)};
  m_reduce_us = std::chrono::duration_cast<us_t>(now - m_t0[instance]).count();
}
