#include "hip/hip_runtime.h"
#include "NoOpReducer.hh"

#include "GpuAsyncLib.hh"
#include "MemPool.hh"
#include "Detector.hh"
#include "drp/drp.hh"
#include "xtcdata/xtc/VarDef.hh"
#include "xtcdata/xtc/DescData.hh"
#include "psalg/utils/SysLog.hh"

using logging = psalg::SysLog;
using namespace XtcData;
using namespace Drp::Gpu;

namespace Drp {
  namespace Gpu {

class NoOpReducerDef : public VarDef
{
public:
  enum index { noOp };

  NoOpReducerDef()
  {
    NameVec.push_back({"noOp", Name::UINT8, 1});
  }
};

  } // Gpu
} // Drp


NoOpReducer::NoOpReducer(const Parameters& para, const MemPoolGpu& pool, Detector& det) :
  ReducerAlgo(para, pool, det)
{
}

// GPU kernel for actually performing the data reduction
// In this case, the calibrated data is just copied to the output buffer
static __global__ void _noOpReduce(const unsigned&                 index,
                                   float const* const __restrict__ calibBuffers,
                                   const size_t                    calibBufsCnt,
                                   uint8_t    * const __restrict__ dataBuffers,
                                   const size_t                    dataBufsCnt)
{
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  float const* const __restrict__ calib = &calibBuffers[index * calibBufsCnt];
  float*       const __restrict__ data  = (float*)(&dataBuffers[index * dataBufsCnt]);
  for (unsigned i = offset; i < calibBufsCnt; i += stride) {
    data[i] = calib[i];
  }

  // Place the size of the reduced data just before the data
  if (offset == 0) {
    size_t* const __restrict__ extent = &((size_t*)data)[-1];
    *extent = calibBufsCnt * sizeof(*calib); //Buffers);
  }
}

// This routine records the graph that does the data reduction
void NoOpReducer::recordGraph(hipStream_t       stream,
                              const unsigned&    index,
                              float const* const calibBuffers,
                              const size_t       calibBufsCnt,
                              uint8_t    * const dataBuffers,
                              const size_t       dataBufsCnt)
{
  int threads = 1024;
  int blocks  = (calibBufsCnt + threads-1) / threads; // @todo: Limit this?
  _noOpReduce<<<blocks, threads, 0, stream>>>(index,
                                              calibBuffers,
                                              calibBufsCnt,
                                              dataBuffers,
                                              dataBufsCnt);
}

void NoOpReducer::reduce(hipGraphExec_t graph, hipStream_t stream, unsigned index, size_t* dataSize)
{
  chkFatal(hipGraphLaunch(graph, stream));

  auto maxSize = m_pool.reduceBufsReserved() + m_pool.reduceBufsSize();
  auto buffer  = &m_pool.reduceBuffers_d()[index * maxSize];
  auto pSize   = buffer - sizeof(*dataSize);
  chkError(hipMemcpyAsync((void*)dataSize, pSize, sizeof(*dataSize), hipMemcpyDeviceToHost, stream));
}

unsigned NoOpReducer::configure(Xtc& xtc, const void* bufEnd)
{
  // Set up the names for L1Accept data
  Alg alg("noOp", 0, 0, 0);
  NamesId namesId(m_det.nodeId, ReducerNamesIndex);
  Names& names = *new(xtc, bufEnd) Names(bufEnd,
                                         m_para.detName.c_str(), alg,
                                         m_para.detType.c_str(), m_para.serNo.c_str(), namesId, m_para.detSegment);
  NoOpReducerDef reducerDef;
  names.add(xtc, bufEnd, reducerDef);
  m_det.namesLookup()[namesId] = NameIndex(names);

 return 0;
}

void NoOpReducer::event(Xtc& xtc, const void* bufEnd, unsigned dataSize)
{
  // The Xtc header is constructed in the CPU's pebble buffer, but this buffer
  // is not used to hold all of the data.  However, bufEnd has to point to a
  // location that makes it appear that the buffer is large enough to contain
  // both the header and data so that the Xtc allocate in data.set_array_shape()
  // can succeed.  This may be larger than the pebble buffer and we therefore
  // must be careful not to write beyond its end.
  //printf("*** NoOpReducer event: xtc %p, extent %u, size %u", &xtc, xtc.extent, dataSize);

  // Data is Reduced data
  NamesId namesId(m_det.nodeId, ReducerNamesIndex);

  // CreateData places into the Xtc, in one contiguous block:
  // - the ShapesData Xtc
  // - the Shapes Xtc with its payload
  // - the Data Xtc (the payload of which is on the GPU)
  CreateData data(xtc, bufEnd, m_det.namesLookup(), namesId);

  // Update the header with the size and shape of the data payload.
  // This does not write beyond the Xtc header in the pebble buffer.
  unsigned dataShape[MaxRank] = { dataSize };
  data.set_array_shape(NoOpReducerDef::noOp, dataShape);
}

// The class factory

extern "C" Drp::Gpu::ReducerAlgo* createReducer(const Drp::Parameters&      para,
                                                const Drp::Gpu::MemPoolGpu& pool,
                                                Drp::Gpu::Detector&         det)
{
  return new Drp::Gpu::NoOpReducer(para, pool, det);
}
