#include "hip/hip_runtime.h"
#include "GpuWorker_impl.hh"

#include <GpuAsync.h>
#include "drp.hh"
// @todo: Revisit: #include "GpuDetector.hh"
#include "Detector.hh"
#include "spscqueue.hh"

#include <hip/hip_runtime.h>

#include "psalg/utils/SysLog.hh"
#include "psdaq/service/EbDgram.hh"
#include "psdaq/service/kwargs.hh"

using namespace Drp;
using namespace Pds;
using namespace XtcData;
using logging = psalg::SysLog;


#define EMPTY ""           // Ensures there is an arg when __VA_ARGS__ is blank
#define chkFatal(rc, ...)  checkError((rc), #rc, __FILE__, __LINE__, true,  EMPTY __VA_ARGS__)
#define chkError(rc, ...)  checkError((rc), #rc, __FILE__, __LINE__, false, EMPTY __VA_ARGS__)

namespace Drp {

static
bool checkError(hipError_t status, const char* func, const char* file, int line, bool crash=true, const char* msg="")
{
  if (status != hipSuccess) {
    const char* perrstr = 0;
    hipError_t    rc      = hipDrvGetErrorString(status, &perrstr);
    if (rc == hipSuccess) {
      if (perrstr) {
        logging::error("%s:%d:\n  '%s'\n  status %d: info: %s - %s\n", file, line, func, status, perrstr, msg);
      } else {
        logging::error("%s:%d:\n  '%s'\n  status %d: info: unknown error - %s\n", file, line, func, status, msg);
      }
    } else {
      logging::error("%s:%d:\n  '%s'\n  status %d: info: unknown error - %s\n", file, line, func, status, msg);
    }
    if (crash)  abort();
    return true;
  }
  return false;
}

static
bool checkError(hipError_t status, const char* func, const char* file, int line, bool crash=true, const char* msg="")
{
  if (status != hipSuccess) {
    logging::error("%s:%d:  '%s'\n  %s\n  status %d: info: %s - %s\n", file, line, func, status, hipGetErrorString(status), msg);
    if (crash)  abort();
    return true;
  }
  return false;
}

struct DmaDsc
{
  int32_t  ret;
  uint32_t size;                        // @todo: Guess
  uint32_t index;
  uint32_t dest;
  uint32_t flags;
  uint32_t errors;
  uint32_t _rsvd[2];                    // @todo: ???
};

};

CudaContext::CudaContext()
{
  logging::info("CudaContext constructed in process ID %lu", syscall(SYS_gettid));

  chkFatal(hipInit(0), "Error while initting cuda");
}

bool CudaContext::initialize(int device)
{
  logging::info("CudaContext initialized in process ID %lu", syscall(SYS_gettid));

  int devs = 0;
  if (chkError(hipGetDeviceCount(&devs)))
    return false;
  logging::debug("Total GPU devices %d\n", devs);
  if (devs <= 0) {
    logging::error("No GPU devices available!\n");
    return false;
  }

  device = device < 0 ? 0 : device;
  if (devs <= device) {
    logging::error("Invalid GPU device number %d! There are only %d devices available\n", device, devs);
    return false;
  }

  // Actually get the device...
  if (chkError(hipDeviceGet(&m_device, device), "Could not get GPU device!"))
    return false;

  // Spew device name
  char name[256];
  if (chkError(hipDeviceGetName(name, sizeof(name), m_device)))
    return false;
  logging::debug("Selected GPU device: %s\n", name);

  // Set required attributes
  int res;
  if (chkError(hipDeviceGetAttribute(&res, CU_DEVICE_ATTRIBUTE_CAN_USE_STREAM_MEM_OPS_V1, m_device)))
    return false;
  if (!res) {
    logging::warning("This device does not support CUDA Stream Operations, this code will not run!\n");
    logging::error(
                   "  Consider setting NVreg_EnableStreamMemOPs=1 when loading the NVIDIA kernel module, "
                   "if your GPU is supported.\n");
    return false;
  }

  // Report memory totals
  size_t global_mem = 0;
  if (chkError(hipDeviceTotalMem(&global_mem, m_device)))
    return false;
  logging::debug("Global memory: %zu MB\n", global_mem >> 20);
  if (global_mem > (size_t)4 << 30)
    logging::debug("64-bit Memory Address support\n");

  int value;
  if (chkError(hipDeviceGetAttribute(&value, hipDeviceAttributeUnifiedAddressing, m_device)))
    return false;
  logging::debug("Device supports unified addressing: %s\n", value ? "YES" : "NO");

  // Create context
  if (chkError(hipCtxCreate(&m_context, 0, m_device)))
    return false;

  return true;
}


void CudaContext::listDevices() {
  int devs = 0;
  if (chkError(hipGetDeviceCount(&devs), "Unable to get device count"))
    return;

  for (int i = 0; i < devs; ++i) {
    hipDevice_t dev;
    if (chkError(hipDeviceGet(&dev, i))) {
      logging::error("Unable to get device %d", i);
      continue;
    }
    char name[256];
    if (chkError(hipDeviceGetName(name, sizeof(name), dev)))
      break;
    logging::info("%d: %s\n", i, name);
  }
}


__global__ void workerFunc(unsigned last, hipDeviceptr_t in, /*Batch* batch,*/ bool& full, bool& sawDisable)
{
#if 0
  // @todo: Process input timing header into a batch entry
  //        Include the PGPEvent information in the entry structure
  // @todo: Batch is full when Disable is seen

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int end = last + (uint32_t*)in[1];
  if (end >= batchMaxIdx) {
    end  = batchMaxIdx;
    full = true;
  }
  printf("*** idx %d, last %d, end %d\n", idx, last, end);
  if (idx >= last && idx <= end) {
    DmaDsc* dmas = (DmaDsc*)(in + 2*sizeof(uint32_t));
    DmaDsc& dma = dmas[idx];

    uint32_t size = dma.ret;
    uint32_t index = dma.index;
    uint32_t lane = (dma.dest >> 8) & 7;
    m_dmaSize = size;
    m_dmaBytes += size;
    // dmaReadBulkIndex() returns a maximum size of m_pool.dmaSize(), never larger.
    // If the DMA overflowed the buffer, the excess is returned in a 2nd DMA buffer,
    // which thus won't have the expected header.  Take the exact match as an overflow indicator.
    if (size == m_pool.dmaSize()) {
        // @todo: Revisit how to handle/return errors
        logging::critical("DMA overflowed buffer: %d vs %d", size, m_pool.dmaSize());
        abort();
    }

    const TimingHeader* timingHeader = det.getTimingHeader(index);
    uint32_t evtCounter = timingHeader->evtCounter & 0xffffff;
    uint32_t pgpIndex = evtCounter & (m_pool.nDmaBuffers() - 1);
    PGPEvent* event = &m_pgpEvents[pgpIndex - last]; // @todo: There are only batchSize pgpEvents in this pool?
    DmaBuffer* buffer = &event->buffers[lane];
    buffer->size = size;
    buffer->index = index;
    event->mask |= (1 << lane);

    m_pool.countDma(); // DMA buffer was allocated when f/w incremented evtCounter

    // Copy the TimingHeader to the host using managed memory
    // @todo: Fix TimingHeader class to allow copying
    m_timingHeaders[index] = *timingHeader;

    TransitionId::Value transitionId = timingHeader->service();

    // Process data payload
    if (transitionId == TransitionId::L1Accept) {
      m_det.event(*timingHeader, event); // @todo: Needs an output buffer
    } else if (transitionId == TransitionId::SlowUpdate) {
      m_det.slowupdate(*timingHeader);   // @todo: Does this produce output or just update state?
    } else {                             // @todo: Check/assert for non-Disable transitions?
      sawDisable |= transitionId == TransitionId::Disable;
    }
  }
#else
  //unsigned batchMaxIdx = 1;             // @todo: For now
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int end = -1; //last + ((uint32_t*)in)[1];
  //if (end >= batchMaxIdx) {
  //  end  = batchMaxIdx;
  //  full = true;
  //}
  printf("*** thr %d, dim %d, bIdx %d, idx %d, last %d, end %d\n",
         threadIdx.x, blockDim.x, blockIdx.x, idx, last, end);
  //if (idx >= last && idx <= end) {
    uint32_t* hdr = (uint32_t*)in;
    printf("*** idx %4d  hdr: %08x %08x %08x %08x  %08x %08x %08x %08x\n",
           idx, hdr[0], hdr[1], hdr[2], hdr[3], hdr[4], hdr[5], hdr[6], hdr[7]);
    auto th = (uint32_t*)&hdr[8];
    full = true;
    auto ctlPid = *(uint64_t*)&th[0];
    auto pid = ctlPid & 0x00ffffffffffffff;
    auto ctl = (ctlPid >> 56) & 0xff;
    auto svc = ctl & 0xf;
    auto ts  = *(uint64_t*)&th[2];
    auto env = th[3];
    auto ctr = th[4];
    auto opq = &th[5];
    sawDisable = svc == TransitionId::Disable;
    printf("*** idx %4d:  pid %016lx, ts %016lx, env %08x, ctr %08x, opq %08x %08x\n",
           idx, pid, ts, env, ctr, opq[0], opq[1]);
    //}
#endif
}


GpuMemPool::GpuMemPool(const Parameters& para, MemPool& pool) :
  dmaBuffers(MAX_BUFFERS), // pool.nbuffers()), // @todo: Revisit nbuffs and size
  m_pool    (pool)
{
  logging::info("GpuMemPool constructed in process ID %lu", syscall(SYS_gettid));
}

GpuMemPool::~GpuMemPool()
{
  for (hipDeviceptr_t& dmaBuffer : dmaBuffers) {
    _gpuUnmapFpgaMem(dmaBuffer);
  }
  dmaBuffers.clear();

  ssize_t rc;
  if ((rc = gpuRemNvidiaMemory(fd())) < 0)
    logging::error("gpuRemNvidiaMemory failed: %zd: %M", rc);
}

int GpuMemPool::initialize()
{
  // Clear out any left-overs from last time
  int res = gpuRemNvidiaMemory(fd());
  if (res < 0)  logging::error("Error in gpuRemNvidiaMemory\n");
  logging::debug("Done with gpuRemNvidiaMemory() cleanup\n");

  // Allocate buffers on the GPU
  // This handles allocating buffers on the device and registering them with the driver.
  for (hipDeviceptr_t& dmaBuffer : dmaBuffers) {
    if (_gpuMapFpgaMem(dmaBuffer, 0, dmaSize(), 1) != 0) {
      logging::error("Failed to alloc buffer list at number %zd",
                     &dmaBuffer - &dmaBuffers[0]);
      return -1;
    }
  }
  logging::debug("Done with device mem alloc\n");

  return 0;
}

// To avoid including drp.hh in GpuWorker_impl.hh:
unsigned               GpuMemPool::count()     const { return dmaBuffers.size(); }
size_t                 GpuMemPool::dmaSize()   const { return m_pool.dmaSize(); }
unsigned               GpuMemPool::nbuffers()  const { return m_pool.nbuffers(); }
int                    GpuMemPool::fd()        const { return m_pool.fd(); }
std::vector<PGPEvent>& GpuMemPool::pgpEvents() const { return m_pool.pgpEvents; }
Pebble&                GpuMemPool::pebble()    const { return m_pool.pebble; }
unsigned               GpuMemPool::allocate()        { return m_pool.allocate(); }

int GpuMemPool::_gpuMapFpgaMem(hipDeviceptr_t& buffer, uint64_t offset, size_t size, int write)
{
  auto idx = &buffer - &dmaBuffers[0];

  if (chkError(hipMalloc(&buffer, size))) {
    return -1;
  }
  logging::debug("Done with device mem alloc %zd\n", idx);
  hipMemsetD8(buffer, 0, size);

  int flag = 1;
  // This attribute is required for peer shared memory. It will synchronize every synchronous memory operation on this block of memory.
  if (chkError(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, buffer))) {
    hipFree(buffer);
    return -1;
  }
  logging::debug("Done with set attr %zd\n", idx);

  if (gpuAddNvidiaMemory(fd(), write, buffer, size) < 0) {
    logging::error("gpuAddNvidiaMemory failed for buffer %zd", idx);
    hipFree(buffer);
    return -1;
  }
  logging::debug("Done with gpuAddNvidiaMemory %zd\n", idx);

  return 0;
}

void GpuMemPool::_gpuUnmapFpgaMem(hipDeviceptr_t& buffer)
{
  chkError(hipFree(buffer));

  // FIXME: gpuOnly memory cannot be unmapped?
}


GpuWorker_impl::GpuWorker_impl(const Parameters& para, MemPool& pool, Detector& det) :
  m_det     (det),
  m_pool    (para, pool),
  m_streams (m_pool.count()),
  m_dmaIndex(0),
  m_para    (para)
{
  logging::info("GpuWorker_impl constructed in process ID %lu", syscall(SYS_gettid));

  ////////////////////////////////////////////
  // Setup GPU
  ////////////////////////////////////////////

  if (para.verbose)
    m_context.listDevices();

  unsigned gpuId = 0;
  if (para.kwargs.find("gpuId") != para.kwargs.end())
    gpuId = std::stoul(const_cast<Parameters&>(para).kwargs["gpuId"]);

  if (!m_context.initialize(gpuId)) {
    logging::critical("CUDA initialize failed");
    abort();
  }
  logging::debug("Done with context setup\n");

  ////////////////////////////////////
  // Setup memory
  ////////////////////////////////////

  if (m_pool.initialize() < 0) {
    logging::critical("Error setting up memory");
    abort();
  }
  logging::debug("Done with setting up memory\n");

  ////////////////////////////////////
  // Allocate streams
  ////////////////////////////////////

  /** Allocate a stream per buffer **/
  for (auto& stream : m_streams) {
    chkFatal(hipStreamCreate(&stream), "Error creating streams");
  }
  logging::debug("Done with creating streams\n");
}

void GpuWorker_impl::timingHeaders(unsigned index, TimingHeader* buffer)
{
  auto idx = index & (m_streams.size() - 1);
  chkFatal(hipMemcpyDtoH((void*)buffer, m_pool.dmaBuffers[idx], sizeof(*buffer)));
}

// @todo: This method is called when it has been recognized that data
//        has been DMAed into GPU memory and is ready to be processed
void GpuWorker_impl::process(Batch& batch, bool& sawDisable)
{
  // Set up a buffer pool for timing headers visible to the host
  // memcpy timing headers from device into this host pool
  // memcpy the TEB input data right after the timing header?
  //   Or put them in a separate pool?
  // Form a batch of them
  // Return from this routine when batch is full or Disable is seen

}

// @todo: This method is called to wait for data to be DMAed into GPU memory
//        This method must then do several things:
//        - Find and copy the TimingHeader to a host buffer and share that
//          buffer's index
//        - Do the equivalent of the det.event() and det.slowUpdate() routines
//          to reorganize the data and prepare the Xtc header
//        - Prepare the TEB input data
// @todo: Spread this work across GPU blocks/threads/streams?
// This is called from the GpuDetector::reader() method, which has nothing to
// do until Disable is seen, so no obvious need for worker thread(s)
void GpuWorker_impl::reader(uint32_t start, SPSCQueue<Batch>& collectorGpuQueue)
{
  logging::debug("GpuWorker_impl::reader() running in process ID %lu", syscall(SYS_gettid));

  // Set the context for the current thread
  chkFatal(hipCtxSetCurrent(m_context.context()));
  logging::debug("Done with setting context\n");

  size_t    size          = m_pool.dmaSize();
  uint32_t* hostWriteBuff = (uint32_t*)malloc(size);

  const uint32_t bufferMask = m_pool.nbuffers() - 1;
  Batch          batch{start + 1, 0};

  // Handle L1Accepts, SlowUpdates and Disable
  bool     full       = false;
  bool     sawDisable = false;
  unsigned last       = 0;
  unsigned dmaIndex   = m_dmaIndex;
  unsigned evtCounter;
  do {                                  // @todo: Handle each stream in a separate thread
    auto&        stream     = m_streams[dmaIndex];
    hipDeviceptr_t& hwWritePtr = m_pool.dmaBuffers[dmaIndex];

    // Clear the GPU memory handshake space to zero
    logging::debug("Clear memory\n");
    chkFatal(hipStreamWriteValue32(stream, hwWritePtr + 4, 0x00, 0));

    // Write to the DMA start register in the FPGA
    logging::debug("Trigger write to buffer %d\n", dmaIndex);
    auto rc = gpuSetWriteEn(m_pool.fd(), dmaIndex);
    if (rc < 0) {
      logging::critical("Failed to reenable buffer %d for write: %zd\n", dmaIndex, rc);
      perror("gpuSetWriteEn");
      abort();
    }

    // Spin on the handshake location until the value is greater than or equal to 1
    // This waits for the data to arrive in the GPU before starting the processing
    logging::debug("Wait memory value\n");
    chkFatal(hipStreamWaitValue32(stream, hwWritePtr + 4, 0x1, hipStreamWaitValueGte));
    chkError(hipStreamSynchronize(stream));
    logging::debug("Done waiting\n");

    unsigned nDmaRet = 1;  //*((unsigned*)(hwWritePtr + 4));

    chkError(hipMemcpyDtoH(hostWriteBuff, hwWritePtr, sizeof(DmaDsc)+sizeof(TimingHeader)));
    auto dsc = (DmaDsc*)&hostWriteBuff[0];
    printf("*** hdr: ret %08x,  sz %08x, idx %08x, dst %08x, flg %08x, err %08x, rsvd %08x %08x\n",
           dsc->ret, dsc->size, dsc->index, dsc->dest, dsc->flags, dsc->errors,
           dsc->_rsvd[0], dsc->_rsvd[1]);
    auto th  = (TimingHeader*)&hostWriteBuff[8];
    printf("**G  th: ctl %02x, pid %014lx, ts %016lx, env %08x, ctr %08x, opq %08x %08x\n",
           th->control(), th->pulseId(), th->time.value(), th->env, th->evtCounter,
           th->_opaque[0], th->_opaque[1]);

    // @todo: Need indices, errors, etc., like from dmaBulkReadDmaIndex()
    // @todo: Handle multiple lanes
    uint32_t size = dsc->size;
    uint32_t lane = (dsc->dest >> 8) & 7;
    m_dmaSize   = size;
    m_dmaBytes += size;
    // @todo: Is this the case here also?
    // dmaReadBulkIndex() returns a maximum size of m_pool.dmaSize(), never larger.
    // If the DMA overflowed the buffer, the excess is returned in a 2nd DMA buffer,
    // which thus won't have the expected header.  Take the exact match as an overflow indicator.
    if (size == m_pool.dmaSize()) {
      logging::critical("DMA overflowed buffer: %d vs %d", size, m_pool.dmaSize());
      abort();
    }

    // @todo: dsc->index is always 0?
    //if (dmaIndex != dsc->index)
    //  logging::error("DMA index mismatch: got %u, expected %u\n",
    //                 dsc->index, dmaIndex);
    evtCounter = th->evtCounter & bufferMask;
    if (evtCounter != batch.start + last)
      logging::error("Event counter mismatch: got %u, expected %u\n",
                     evtCounter, batch.start + last);

    sawDisable = th->service() == TransitionId::Disable;

    PGPEvent* event = &m_pool.pgpEvents()[evtCounter];
    event->mask |= (1 << lane);

    // Allocate a pebble buffer once the event is built
    auto counter       = m_pool.allocate(); // This can block
    auto pebbleIndex   = counter & (m_pool.nbuffers() - 1);
    event->pebbleIndex = pebbleIndex;

    // Make a new dgram in the pebble
    // It must be an EbDgram in order to be able to send it to the MEB
    auto dgram = new(m_pool.pebble()[pebbleIndex]) EbDgram(*th, m_det.nodeId, m_para.rogMask);

    // @todo: Process the data to extract TEB input and calibrate.  Also reduce/compress?
    //if (th->service() == TransitionId::L1Accept)
    //  this->event(*th, event);
    //else if (th->service() == TransitionId::SlowUpdate)
    //  this->slowUpdate(*th);

    last       += nDmaRet;
    batch.size += nDmaRet;
    full = batch.size == 4;             // @todo: arbitrary

    printf("*** nDmaRet %d, last %u, size %u, full %d, sawDisable %d\n",
           nDmaRet, last, batch.size, full, sawDisable);

    if (full || sawDisable) {
      // Queue the batch to the Collector
      collectorGpuQueue.push(batch);

      // Reset to the beginning of the batch
      full = false;
      last = 0;
      batch.start = th->evtCounter + 1;
      batch.size = 0;
    }
    dmaIndex = (dmaIndex + 1) & (m_streams.size() - 1);
  } while (!sawDisable);

  // Clean up
  free(hostWriteBuff);
  m_dmaIndex   = dmaIndex;   // Ensure we start with the correct buffer next time
  m_lastEvtCtr = evtCounter; // Ensure PgpReader::handle() doesn't complain about jumps

  logging::debug("Returning from reader\n");
}
